#include "hip/hip_runtime.h"

__device__ static int ros_Integrator_ros3(double * __restrict__ var, const double * __restrict__ fix, const double Tstart, const double Tend, double &T,
        //  Integration parameters
        const int autonomous, const int vectorTol, const int Max_no_steps, 
        const double roundoff, const double Hmin, const double Hmax, const double Hstart, double &Hexit, 
        const double FacMin, const double FacMax, const double FacRej, const double FacSafe, 
        //  Status parameters
        int &Nfun, int &Njac, int &Nstp, int &Nacc, int &Nrej, int &Ndec, int &Nsol, int &Nsng,
        //  cuda global mem buffers              
        const double * __restrict__ rconst,  const double * __restrict__ absTol, const double * __restrict__ relTol, double * __restrict__ varNew, double * __restrict__ Fcn0, 
        double * __restrict__ K, double * __restrict__ dFdT, double * __restrict__ jac0, double * __restrict__ Ghimj, double * __restrict__ varErr,
        // for update_rconst
        const double * __restrict__ khet_st, const double * __restrict__ khet_tr,
        const double * __restrict__ jx,
        // VL_GLO
        const int VL_GLO,
        const int offset)
{
    int index = blockIdx.x*blockDim.x+threadIdx.x+offset;

    double H, Hnew, HC, HC0,HC1, HG, Fac; // Tau - not used
    double Err; //*varErr;
    int direction;
    int rejectLastH, rejectMoreH;
    const double DELTAMIN = 1.0E-5;

    const int ros_S = 3;

    //   ~~~>  Initial preparations
    T = Tstart;
    Hexit = 0.0;
    H = fmin(Hstart,Hmax);
    if (fabs(H) <= 10.0*roundoff) 
        H = DELTAMIN;

    if (Tend  >=  Tstart)
    {
        direction = + 1;
    }
    else
    {
        direction = - 1;
    }

    rejectLastH=0;
    rejectMoreH=0;

    // TimeLoop: 
    while((direction > 0) && ((T- Tend)+ roundoff <= ZERO) || (direction < 0) && ((Tend-T)+ roundoff <= ZERO))
    {
        if (Nstp > Max_no_steps) //  Too many steps
            return -6;
        //  Step size too small
        if (H <= roundoff){  //  Step size too small
            //if (((T+ 0.1*H) == T) || (H <= roundoff)) {
            return -7;
        }

        //   ~~~>  Limit H if necessary to avoid going beyond Tend
        Hexit = H;
        H = fmin(H,fabs(Tend-T));

        //   ~~~>   Compute the function at current time
        Fun(var, fix, rconst, Fcn0, Nfun, VL_GLO);

        //   ~~~>  Compute the function derivative with respect to T
        if (!autonomous)
            ros_FunTimeDerivative(T, roundoff, var, fix, rconst, dFdT, Fcn0, Nfun, khet_st, khet_tr, jx,  VL_GLO); /// VAR READ - fcn0 read

        //   ~~~>   Compute the Jacobian at current time
        Jac_sp(var, fix, rconst, jac0, Njac, VL_GLO);   /// VAR READ 

        //   ~~~>  Repeat step calculation until current step accepted
        // UntilAccepted: 
        while(1)
        {
            ros_PrepareMatrix(H, direction, 0.43586652150845899941601945119356E+00 , jac0, Ghimj, Nsng, Ndec, VL_GLO);

            { // istage=0
                for (int i=0; i<NVAR; i++){
                    K(index,0,i)  = Fcn0(index,i);				// FCN0 Read
                }

                if ((!autonomous))
                {
                    HG = direction*H*0.43586652150845899941601945119356E+00;
                    for (int i=0; i<NVAR; i++){
                        K(index,0,i) += dFdT(index,i)*HG;
		     }
                }
                ros_Solve(Ghimj, K, Nsol, 0, ros_S);
            } // Stage

            {   // istage = 1
                for (int i=0; i<NVAR; i++){		
                    varNew(index,i) = K(index,0,i)  + var(index,i);
                }
                Fun(varNew, fix, rconst, varNew, Nfun,VL_GLO); // FCN <- varNew / not overlap 
                HC = -0.10156171083877702091975600115545E+01/(direction*H);
                for (int i=0; i<NVAR; i++){
                    double tmp = K(index,0,i);
                    K(index,1,i) = tmp*HC + varNew(index,i);
                }
                if ((!autonomous))
                {
                    HG = direction*H*0.24291996454816804366592249683314E+00;
                    for (int i=0; i<NVAR; i++){
                        K(index,1,i) += dFdT(index,i)*HG;
		     }
                }
		//	   R   ,RW, RW,  R,        R 
                ros_Solve(Ghimj, K, Nsol, 1, ros_S);
            } // Stage

            {
                int istage = 2;

                HC0 = 0.40759956452537699824805835358067E+01/(direction*H);
                HC1 = 0.92076794298330791242156818474003E+01/(direction*H);

                for (int i=0; i<NVAR; i++){
                    K(index,2,i) = K(index,1,i)*HC1 +   K(index,0,i)*HC0 +  varNew(index,i);
                }
                if ((!autonomous) )
                {
                    HG = direction*H*0.21851380027664058511513169485832E+01;
                    for (int i=0; i<NVAR; i++){
                        K(index,istage,i) += dFdT(index,i)*HG;
		     }
                }
                ros_Solve(Ghimj, K, Nsol, istage, ros_S);
            } // Stage

            //  ~~~>  Compute the new solution
	    for (int i=0; i<NVAR; i++){
                    varNew(index,i) = K(index,0,i)   + K(index,1,i)*0.61697947043828245592553615689730E+01 + K(index,2,i)*(-0.42772256543218573326238373806514) + var(index,i) ;
                    varErr(index,i) = K(index,0,i)/2 + K(index,1,i)*(-0.29079558716805469821718236208017E+01) + K(index,2,i)*(0.22354069897811569627360909276199);
	    }

            Err = ros_ErrorNorm(var, varNew, varErr, absTol, relTol, vectorTol);   

//  ~~~> New step size is bounded by FacMin <= Hnew/H <= FacMax
            Fac  = fmin(FacMax,fmax(FacMin,FacSafe/pow(Err,ONE/3.0)));
            Hnew = H*Fac;

//  ~~~>  Check the error magnitude and adjust step size
            Nstp = Nstp+ 1;
            if((Err <= ONE) || (H <= Hmin)) // ~~~> Accept step
            {
                Nacc = Nacc + 1;
                for (int j=0; j<NVAR ; j++)
                    var(index,j) =  fmax(varNew(index,j),ZERO);  /////////// VAR WRITE - last VarNew read

                T = T +  direction*H;
                Hnew = fmax(Hmin,fmin(Hnew,Hmax));
                if (rejectLastH)   // No step size increase after a rejected step
                    Hnew = fmin(Hnew,H);
                rejectLastH = 0;
                rejectMoreH = 0;
                H = Hnew;

            	break;  //  EXIT THE LOOP: WHILE STEP NOT ACCEPTED
            }
            else      // ~~~> Reject step
            {
                if (rejectMoreH)
                    Hnew = H*FacRej;
                rejectMoreH = rejectLastH;
                rejectLastH = 1;
                H = Hnew;
                if (Nacc >= 1)
                    Nrej += 1;
            } //  Err <= 1
        } // UntilAccepted
    } // TimeLoop
//  ~~~> Succesful exit
    return 0; //  ~~~> The integration was successful
}

__global__ 
void Rosenbrock_ros3(double * __restrict__ conc, const double Tstart, const double Tend, double * __restrict__ rstatus, int * __restrict__ istatus,
                const int autonomous, const int vectorTol, const int UplimTol, const int Max_no_steps,
                double * __restrict__ d_jac0, double * __restrict__ d_Ghimj, double * __restrict__ d_varNew, double * __restrict__ d_K, double * __restrict__ d_varErr,double * __restrict__ d_dFdT ,double * __restrict__ d_Fcn0, double * __restrict__ d_var, double * __restrict__ d_fix, double * __restrict__ d_rconst,
                const double Hmin, const double Hmax, const double Hstart, const double FacMin, const double FacMax, const double FacRej, const double FacSafe, const double roundoff,
                const double * __restrict__ absTol, const double * __restrict__ relTol,
    	        const double * __restrict__ khet_st, const double * __restrict__ khet_tr,
		const double * __restrict__ jx,
                const double * __restrict__ temp_gpu,
                const double * __restrict__ press_gpu,
                const double * __restrict__ cair_gpu,
                const int VL_GLO, const int offset)
{
    int index = blockIdx.x*blockDim.x+threadIdx.x + offset;


    /* 
     *  In theory someone can aggregate accesses together,
     *  however due to algorithm, threads access 
     *  different parts of memory, making it harder to
     *  optimize accesses. 
     *
     */
    double *Ghimj  = &d_Ghimj[index*LU_NONZERO];    
    double *K      = &d_K[index*NVAR*3];
    double *varNew = &d_varNew[index*NVAR];
    double *Fcn0   = &d_Fcn0[index*NVAR];
    double *dFdT   = &d_dFdT[index*NVAR];
    double *jac0   = &d_jac0[index*LU_NONZERO];
    double *varErr = &d_varErr[index*NVAR];
    double *var    = &d_var[index*NSPEC];
    double *fix    = &d_fix[index*NFIX];
    double *rconst = &d_rconst[index*NREACT];

    const int method = 2;

    if (index < VL_GLO)
    {

        int Nfun,Njac,Nstp,Nacc,Nrej,Ndec,Nsol,Nsng;
        double Texit, Hexit;

        Nfun = 0;
        Njac = 0;
        Nstp = 0;
        Nacc = 0;
        Nrej = 0;
        Ndec = 0;
        Nsol = 0;
        Nsng = 0;


        /* Copy data from global memory to temporary array */
        /*
         * Optimization note: if we ever have enough constant
         * memory, we could use it for storing the data.
         * In current architectures if we use constant memory
         * only a few threads will be able to run on the fly.
         *
         */
        for (int i=0; i<NSPEC; i++)
            var(index,i) = conc(index,i);

        for (int i=0; i<NFIX; i++)
            fix(index,i) = conc(index,NVAR+i);

        update_rconst(var, khet_st, khet_tr, jx, rconst, temp_gpu, press_gpu, cair_gpu, VL_GLO, offset); 

        ros_Integrator_ros3(var, fix, Tstart, Tend, Texit,
                //  Integration parameters
                autonomous, vectorTol, Max_no_steps, 
                roundoff, Hmin, Hmax, Hstart, Hexit, 
                FacMin, FacMax, FacRej, FacSafe,
                //  Status parameters
                Nfun, Njac, Nstp, Nacc, Nrej, Ndec, Nsol, Nsng,
                //  cuda global mem buffers              
                rconst, absTol, relTol, varNew, Fcn0,  
                K, dFdT, jac0, Ghimj,  varErr, 
                // For update rconst
                khet_st, khet_tr, jx,
                VL_GLO, offset
                );

        for (int i=0; i<NVAR; i++)
            conc(index,i) = var(index,i); 


        /* Statistics */
        istatus(index,ifun) = Nfun;
        istatus(index,ijac) = Njac;
        istatus(index,istp) = Nstp;
        istatus(index,iacc) = Nacc;
        istatus(index,irej) = Nrej;
        istatus(index,idec) = Ndec;
        istatus(index,isol) = Nsol;
        istatus(index,isng) = Nsng;
        // Last T and H
        rstatus(index,itexit) = Texit;
        rstatus(index,ihexit) = Hexit; 
    }
}




