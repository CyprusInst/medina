#include "hip/hip_runtime.h"
/*************************************************************
 *
 *    kpp_integrate_cuda_prototype.cu
 *    Prototype file for kpp CUDA kernel
 *
 *    Copyright 2016 The Cyprus Institute
 *
 *    Developers: Michail Alvanos - m.alvanos@cyi.ac.cy
 *                Giannis Ashiotis
 *                Theodoros Christoudias - christoudias@cyi.ac.cy
 *
 ********************************************************************/

#include <stdio.h>
#include <unistd.h>
#include "hip/hip_runtime.h"

=#=#=#=#=#=#=#=#=#=#=defines_vars_2=#=#=#=#=#=#=#=#=#=#=

#define BLOCKSIZE 64

//#define MAX_VL_GLO 12288 /* elements that will pass in each call */

#define REDUCTION_SIZE_1 64
#define REDUCTION_SIZE_2 32

=#=#=#=#=#=#=#=#=#=#=defines_vars_1=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=defines_ind_1=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=defines_ind_2=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=defines_ind_3=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=defines_ind_4=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=defines_ind_5=#=#=#=#=#=#=#=#=#=#=

#define ifun 0
#define ijac 1
#define istp 2
#define iacc 3
#define irej 4
#define idec 5
#define isol 6
#define isng 7
#define itexit 0
#define ihexit 1

#define ZERO 0.0
#define ONE 1.0
#define HALF 0.5


/*
 * Fortran to C macros 
 * GPU-friendly array deffinition 
 * i:VL_GLO, j:NVAR 
 *
 */
#define conc(i,j)    conc[(j)*VL_GLO+(i)]
#define khet_st(i,j) khet_st[(j)*VL_GLO+(i)]
#define khet_tr(i,j) khet_tr[(j)*VL_GLO+(i)]
#define jx(i,j)      jx[j*VL_GLO+i]
#define istatus(i,j) istatus[(j)*(VL_GLO)+(i)]
#define rstatus(i,j) rstatus[(j)*(VL_GLO)+(i)]


#define ROUND128(X)  (X + (128 - 1)) & ~(128 - 1)

#define rconst(i,j)  rconst[(j)]


/* Temporary arrays allocated in stack */
#define var(i,j)     var[(j)]
#define fix(i,j)     fix[(j)]
#define jcb(i,j)     jcb[(j)]
#define varDot(i,j)  varDot[j]
#define varNew(i,j) varNew[(j)]
#define Fcn0(i,j)   Fcn0[(j)]
#define Fcn(i,j)    Fcn[(j)]
#define Fcn(i,j)    Fcn[(j)]
#define dFdT(i,j)   dFdT[(j)]
#define varErr(i,j) varErr[(j)]
#define K(i,j,k) K[(j)*(NVAR)+(k)]
#define jac0(i,j)    jac0[(j)]
#define Ghimj(i,j)   Ghimj[(j)]



#ifdef DEBUG
#define GPU_DEBUG()\
    gpuErrchk( hipPeekAtLastError()   ); \
    gpuErrchk( hipDeviceSynchronize() ); 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#else 
/* If debug flags are disabled */
#define GPU_DEBUG()
#define gpuErrchk(ans) ans
#endif

/** prefetches into L1 cache */
__device__ inline void prefetch_gl1(const void *p) {
#if __CUDA_ARCH__ <= 300
        asm("prefetch.global.L1 [%0];": :"l"(p));
#endif
}
__device__ inline void prefetch_ll1(const void *p) {
#if __CUDA_ARCH__ <= 300
        asm("prefetch.local.L1 [%0];": :"l"(p));
#endif
}

/** prefetches into L2 cache */
__device__ inline void prefetch_gl2(const void *p) {
#if __CUDA_ARCH__ <= 300
        asm("prefetch.global.L2 [%0];": :"l"(p));
#endif
}
__device__ inline void prefetch_ll2(const void *p) {
#if __CUDA_ARCH__ <= 300
        asm("prefetch.local.L2 [%0];": :"l"(p));
#endif
}



__device__ void  update_rconst(const double * __restrict__ var,
			       const double * __restrict__ khet_st, const double * __restrict__ khet_tr,
			       const double * __restrict__ jx, double * __restrict__ rconst,
 			       const double * __restrict__ temp_gpu,
 			       const double * __restrict__ press_gpu,
 			       const double * __restrict__ cair_gpu,
			       const int VL_GLO, const int offset);

/* This runs on CPU */
double machine_eps_flt()
{
    double machEps = 1.0f;

    do
    {
        machEps /= 2.0f;
        // If next epsilon yields 1, then break, because current
        // epsilon is the machine epsilon.
    }
    while ((double)(1.0 + (machEps/2.0)) != 1.0);

    return machEps;
}

/* This runs on GPU */
__device__ double machine_eps_flt_cuda() 
{
    typedef union 
    {
        long  i64;
        double f64;
    } flt_64;

    flt_64 s;

    s.f64 = 1.;
    s.i64++;
    return (s.f64 - 1.);
}

__device__  static double alpha_AN(const int n, const int ro2type, const double temp, const double cair){
    double alpha=2.E-22, beta=1.0, Yinf_298K=0.43,  F=0.41, m0=0., minf=8.0;
    double Y0_298K, Y0_298K_tp, Yinf_298K_t, zeta, k_ratio, alpha_a;
    /*  IF (ro2type = 1) THEN   m = 0.4                !   primary RO2
        ELSE IF (ro2type = 2) THEN  m = 1.                 !   secondary RO2
        ELSE IF (ro2type = 3) THEN  m = 0.3                !   tertiary RO2
        ELSE  m = 1.
  */
    double m = 1.;
    Y0_298K     = alpha*exp(beta*n);
    Y0_298K_tp  = Y0_298K *cair *pow((temp/298.),(- m0));
    Yinf_298K_t = Yinf_298K * pow((temp/298.),(- minf));
    zeta        = 1/(1+ pow(log10(Y0_298K_tp/Yinf_298K_t),2));
    k_ratio     = (Y0_298K_tp/(1+ Y0_298K_tp/Yinf_298K_t))*pow(F,zeta);
    alpha_a    = k_ratio/(1+ k_ratio) *m;
    return alpha_a;
}
__device__  static double alpha_AN(const int n, const int ro2type, const int bcarb, const int gcarb, const int abic, const double temp, const double cair){
    double alpha=2.E-22, beta=1.0, Yinf_298K=0.43,  F=0.41, m0=0., minf=8.0;
    double Y0_298K, Y0_298K_tp, Yinf_298K_t, zeta, k_ratio, alpha_a;
    double bcf=1., gcf=1., abf=1.;
    double m = 1.; //According to Teng, ref3189

if (bcarb == 1) { bcf = 0.19; }// derived from Praske, ref3190: alpha_AN = 0.03 for the secondary HMKO2 relative to alpha_AN for 6C RO2 (0.16)
if (gcarb == 1) {gcf = 0.44; }// derived from Praske, ref3190: alpha_AN = 0.07 for the primary HMKO2 relative to alpha_AN for 6C RO2 (0.16)
if (abic == 1) { abf = 0.24; }// derived from the ratio of AN- yield for toluene from Elrod et al. (ref3180), 5.5 0x1.9206e69676542p+ 229t & 
                              // 200 torr, and this SAR for linear alkyl RO2 with 9 heavy atoms, 23.3%

    Y0_298K     = alpha*exp(beta*n);
    Y0_298K_tp  = Y0_298K *cair *pow((temp/298.),(- m0));
    Yinf_298K_t = Yinf_298K * pow((temp/298.),(- minf));
    zeta        = 1/(1+ pow(log10(Y0_298K_tp/Yinf_298K_t),2));
    k_ratio     = (Y0_298K_tp/(1+ Y0_298K_tp/Yinf_298K_t))*pow(F,zeta);
    alpha_a    = k_ratio/(1+ k_ratio) *m*bcf*gcf*abf;
    return alpha_a;
}
__device__  static double k_RO2_HO2(const double temp, const int nC){
    return 2.91e-13*exp(1300./temp)*(1.-exp(-0.245*nC)); // ref1630
}
__device__ double ros_ErrorNorm(double * __restrict__ var, double * __restrict__ varNew, double * __restrict__ varErr, 
                                const double * __restrict__ absTol, const double * __restrict__ relTol,
                                const int vectorTol )
{
    double err, scale, varMax;


    err = ZERO;

    if (vectorTol){
        for (int i=0;i<NVAR - 16;i+=16){
            prefetch_ll1(&varErr[i]);
            prefetch_ll1(&absTol[i]);
            prefetch_ll1(&relTol[i]);
            prefetch_ll1(&var[i]);
            prefetch_ll1(&varNew[i]);
        }

        for (int i=0; i<NVAR; i++)
        {
            varMax = fmax(fabs(var[i]),fabs(varNew[i]));
            scale = absTol[i]+ relTol[i]*varMax;

            err += pow((double)varErr[i]/scale,2.0);
        }
        err  = sqrt((double) err/NVAR);
    }else{
        for (int i=0;i<NVAR - 16;i+=16){
            prefetch_ll1(&varErr[i]);
            prefetch_ll1(&var[i]);
            prefetch_ll1(&varNew[i]);
        }

        for (int i=0; i<NVAR; i++)
        {
            varMax = fmax(fabs(var[i]),fabs(varNew[i]));

            scale = absTol[0]+ relTol[0]*varMax;
            err += pow((double)varErr[i]/scale,2.0);
        }
        err  = sqrt((double) err/NVAR);
    }

    return err;


}

=#=#=#=#=#=#=#=#=#=#=kppSolve=#=#=#=#=#=#=#=#=#=#=

__device__ void ros_Solve(double * __restrict__ Ghimj, double * __restrict__ K, int &Nsol, const int istage, const int ros_S)
{

    #pragma unroll 4 
    for (int i=0;i<LU_NONZERO-16;i+=16){
        prefetch_ll1(&Ghimj[i]);
    }

    kppSolve(Ghimj, K, istage, ros_S);
    Nsol++;
}

=#=#=#=#=#=#=#=#=#=#=kppDecomp=#=#=#=#=#=#=#=#=#=#=

__device__ void ros_Decomp(double * __restrict__ Ghimj, int &Ndec, int VL_GLO)
{
    kppDecomp(Ghimj, VL_GLO);
    Ndec++;
}


=#=#=#=#=#=#=#=#=#=#=ros_PrepareMatrix=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=Jac_sp=#=#=#=#=#=#=#=#=#=#=

=#=#=#=#=#=#=#=#=#=#=Fun=#=#=#=#=#=#=#=#=#=#=

__device__ void ros_FunTimeDerivative(const double T, double roundoff, double * __restrict__ var, const double * __restrict__ fix, 
                                      const double * __restrict__ rconst, double *dFdT, double *Fcn0, int &Nfun, 
                                      const double * __restrict__ khet_st, const double * __restrict__ khet_tr,
                                      const double * __restrict__ jx,
                                      const int VL_GLO)
{
    const double DELTAMIN = 1.0E-6;
    double delta,one_over_delta;

    delta = sqrt(roundoff)*fmax(DELTAMIN,fabs(T));
    one_over_delta = 1.0/delta;

    Fun(var, fix, rconst, dFdT, Nfun, VL_GLO);

    for (int i=0; i < NVAR; i++){
        dFdT(index,i) = (dFdT(index,i) - Fcn0(index,i)) * one_over_delta;
    }
}

__device__  static  int ros_Integrator(double * __restrict__ var, const double * __restrict__ fix, const double Tstart, const double Tend, double &T,
        //  Rosenbrock method coefficients
        const int ros_S, const double * __restrict__ ros_M, const double * __restrict__ ros_E, const double * __restrict__ ros_A, const double * __restrict__  ros_C, 
        const double * __restrict__ ros_Alpha, const double * __restrict__ ros_Gamma, const double ros_ELO, const int * ros_NewF, 
        //  Integration parameters
        const int autonomous, const int vectorTol, const int Max_no_steps, 
        const double roundoff, const double Hmin, const double Hmax, const double Hstart, double &Hexit, 
        const double FacMin, const double FacMax, const double FacRej, const double FacSafe, 
        //  Status parameters
        int &Nfun, int &Njac, int &Nstp, int &Nacc, int &Nrej, int &Ndec, int &Nsol, int &Nsng,
        //  cuda global mem buffers              
        const double * __restrict__ rconst,  const double * __restrict__ absTol, const double * __restrict__ relTol, double * __restrict__ varNew, double * __restrict__ Fcn0, 
        double * __restrict__ K, double * __restrict__ dFdT, double * __restrict__ jac0, double * __restrict__ Ghimj, double * __restrict__ varErr,
        // for update_rconst
        const double * __restrict__ khet_st, const double * __restrict__ khet_tr,
        const double * __restrict__ jx,
        // VL_GLO
        const int VL_GLO)
{

    double H, Hnew, HC, HG, Fac; // Tau - not used
    double Err; //*varErr;
    int direction;
    int rejectLastH, rejectMoreH;
    const double DELTAMIN = 1.0E-5;

    //   ~~~>  Initial preparations
    T = Tstart;
    Hexit = 0.0;
    H = fmin(Hstart,Hmax);
    if (fabs(H) <= 10.0*roundoff) 
        H = DELTAMIN;

    if (Tend  >=  Tstart)
    {
        direction = + 1;
    }
    else
    {
        direction = - 1;
    }

    rejectLastH=0;
    rejectMoreH=0;



    //   ~~~> Time loop begins below

    // TimeLoop: 
    while((direction > 0) && ((T- Tend)+ roundoff <= ZERO) || (direction < 0) && ((Tend-T)+ roundoff <= ZERO))
    {
        if (Nstp > Max_no_steps) //  Too many steps
            return -6;
        //  Step size too small
        if (H <= roundoff){  //  Step size too small
            //if (((T+ 0.1*H) == T) || (H <= roundoff)) {
            return -7;
        }

        //   ~~~>  Limit H if necessary to avoid going beyond Tend
        Hexit = H;
        H = fmin(H,fabs(Tend-T));

        //   ~~~>   Compute the function at current time
        Fun(var, fix, rconst, Fcn0, Nfun, VL_GLO);	/// VAR READ - Fcn0 Write

        //   ~~~>  Compute the function derivative with respect to T
        if (!autonomous)
            ros_FunTimeDerivative(T, roundoff, var, fix, rconst, dFdT, Fcn0, Nfun, khet_st, khet_tr, jx,  VL_GLO); /// VAR READ - fcn0 read

        //   ~~~>   Compute the Jacobian at current time
        Jac_sp(var, fix, rconst, jac0, Njac, VL_GLO);   /// VAR READ 

        //   ~~~>  Repeat step calculation until current step accepted
        // UntilAccepted: 
        while(1)
        {
            ros_PrepareMatrix(H, direction, ros_Gamma[0], jac0, Ghimj, Nsng, Ndec, VL_GLO);
            //   ~~~>   Compute the stages
            // Stage: 
            for (int istage=0; istage < ros_S; istage++)
            {
                //   For the 1st istage the function has been computed previously
                if (istage == 0)
                {
                    for (int i=0; i<NVAR; i++){
                        varNew(index,i) = Fcn0(index,i);				// FCN0 Read
                    }
                }
                else if(ros_NewF[istage])
                {
                        for (int i=0; i<NVAR; i++){		
                            varNew(index,i) = var(index,i);
                        }

                    for (int j=0; j < (istage); j++){
                        for (int i=0; i<NVAR; i++){		
                            varNew(index,i) = K(index,j,i)*ros_A[(istage)*(istage-1)/2 + j]  + varNew(index,i);
                        }
                    }
                    Fun(varNew, fix, rconst, varNew, Nfun,VL_GLO); // FCN <- varNew / not overlap 
		} 

		for (int i=0; i<NVAR; i++)		
			K(index,istage,i)  = varNew(index,i);

		for (int j=0; j<(istage); j++)
		{
			HC = ros_C[(istage)*(istage-1)/2 + j]/(direction*H);
			for (int i=0; i<NVAR; i++){
				double tmp = K(index,j,i);
				K(index,istage,i) += tmp*HC;
			}
		}

                if ((!autonomous) && (ros_Gamma[istage] ))
                {
                    HG = direction*H*ros_Gamma[istage];
                    for (int i=0; i<NVAR; i++){
                        K(index,istage,i) += dFdT(index,i)*HG;
		     }
                }
		//	   R   ,RW, RW,  R,        R 
                ros_Solve(Ghimj, K, Nsol, istage, ros_S);


            } // Stage

            //  ~~~>  Compute the new solution
	    for (int i=0; i<NVAR; i++){
		    double tmpNew  = var(index,i); 					/// VAR READ
		    double tmpErr  = ZERO;

		    for (int j=0; j<ros_S; j++){
		    	    double tmp = K(index,j,i);

#ifdef DEBUG
			    if (isnan(tmp)){
			    	printf("Solver detected NAN!");
			    	tmp = 0;
			    }
#endif
			    tmpNew += tmp*ros_M[j];
			    tmpErr += tmp*ros_E[j];
		    }
		    varNew(index,i) = tmpNew;			// varNew is killed
		    varErr(index,i) = tmpErr;
	    }

            Err = ros_ErrorNorm(var, varNew, varErr, absTol, relTol, vectorTol);   /// VAR-varNew READ


//  ~~~> New step size is bounded by FacMin <= Hnew/H <= FacMax
            Fac  = fmin(FacMax,fmax(FacMin,FacSafe/pow(Err,ONE/ros_ELO)));
            Hnew = H*Fac;

//  ~~~>  Check the error magnitude and adjust step size
            Nstp = Nstp+ 1;
            if((Err <= ONE) || (H <= Hmin)) // ~~~> Accept step
            {
                Nacc = Nacc + 1;
                for (int j=0; j<NVAR ; j++)
                    var(index,j) =  fmax(varNew(index,j),ZERO);  /////////// VAR WRITE - last VarNew read

                T = T +  direction*H;
                Hnew = fmax(Hmin,fmin(Hnew,Hmax));
                if (rejectLastH)   // No step size increase after a rejected step
                    Hnew = fmin(Hnew,H);
                rejectLastH = 0;
                rejectMoreH = 0;
                H = Hnew;

            	break;  //  EXIT THE LOOP: WHILE STEP NOT ACCEPTED
            }
            else      // ~~~> Reject step
            {
                if (rejectMoreH)
                    Hnew = H*FacRej;
                rejectMoreH = rejectLastH;
                rejectLastH = 1;
                H = Hnew;
                if (Nacc >= 1)
                    Nrej += 1;
            } //  Err <= 1
        } // UntilAccepted
    } // TimeLoop
//  ~~~> Succesful exit
    return 0; //  ~~~> The integration was successful
}

typedef struct {
 double ros_A[15];
 double ros_C[15];
 int   ros_NewF[8];
 double ros_M[6];
 double ros_E[6];
 double ros_Alpha[6];
 double ros_Gamma[6];
 double ros_ELO;
 int    ros_S;
} ros_t;

/*
 * Lookup tables for different ROS for branch elimination. It is much faster in GPU.
 */
__device__ __constant__  ros_t ros[5] = {
    {       
        {.58578643762690495119831127579030,0,0,0,0,0,0,0,0,0,0,0,0,0,0}, /* ros_A */
        {-1.17157287525380990239662255158060,0,0,0,0,0,0,0,0,0,0,0,0,0,0}, /* ros_C */
        {1,1,0,0,0,0,0,0}, /* ros_NewF */
        {.87867965644035742679746691368545,.29289321881345247559915563789515,0,0,0,0}, /* ros_M */
        {.29289321881345247559915563789515,.29289321881345247559915563789515,0,0,0,0}, /* ros_E */
        {0,1.0,0,0,0,0}, /* ros_Alpha */
        {1.70710678118654752440084436210485,-1.70710678118654752440084436210485,0,0,0,0},  /* ros_Gamma */
        2.0, /* ros_ELO */
        2, /* ros_S*/
    }, /* Ros2 */
    {       
        {1.0,1.0,0,0,0,0,0,0,0,0,0,0,0,0,0}, /* ros_A */
        {-0.10156171083877702091975600115545E+01, 0.40759956452537699824805835358067E+01,0.92076794298330791242156818474003E+01,0,0,0,0,0,0,0,0,0,0,0,0}, /* ros_C */
        {1,1,0,0,0,0,0,0}, /* ros_NewF */
        {0.1E+01,0.61697947043828245592553615689730E+01,-0.42772256543218573326238373806514E+00,0,0,0}, /* ros_M */
        {0.5E+00,- 0.29079558716805469821718236208017E+01,0.22354069897811569627360909276199E+00,0,0,0}, /* ros_E */
        {0.0E+00,0.43586652150845899941601945119356E+00,0.43586652150845899941601945119356E+00,0,0,0}, /* ros_Alpha */
        {0.43586652150845899941601945119356E+00,0.24291996454816804366592249683314E+00,0.21851380027664058511513169485832E+01,0,0,0},  /* ros_Gamma */
        3.0, /* ros_ELO */
        3
    }, /* Ros3 */
    {       
        {0.2000000000000000E+01, 0.1867943637803922E+01, 0.2344449711399156E+00, 0.1867943637803922E+01, 0.2344449711399156E+00,0,0,0,0,0,0,0,0,0,0}, /* ros_A */
        {-0.7137615036412310E+01,0.2580708087951457E+01,0.6515950076447975E+00, - 0.2137148994382534E+01, - 0.3214669691237626E+00, - 0.6949742501781779E+00 ,0,0,0,0,0,0,0,0,0}, /* ros_C */
        {1,1,1,0,0,0,0,0}, /* ros_NewF */
        {0.2255570073418735E+01, 0.2870493262186792E+00, 0.4353179431840180E+00, 0.1093502252409163E+01,0,0}, /* ros_M */
        { -0.2815431932141155E+00, -0.7276199124938920E-01, -0.1082196201495311E+00, -0.1093502252409163E+01, 0, 0}, /* ros_E */
        {0.0, 0.1145640000000000E+01, 0.6552168638155900E+00, 0.6552168638155900E+00,0,0}, /* ros_Alpha */
        { 0.5728200000000000E+00, -0.1769193891319233E+01, 0.7592633437920482E+00, -0.1049021087100450E+00,0,0},  /* ros_Gamma */
        4.0, /* ros_ELO */
        4
    }, /* Ros4 */
    {       
        { 0.0E+00, 2.0E+00, 0.0E+00, 2.0E+00, 0.0E+00, 1.0E+00, 0,0,0,0,0,0,0,0,0}, /* ros_A */
        { 4.0E+00, 1.0E+00, - 1.0E+00,  1.0E+00, - 1.0E+00, - 2.66666666666666666666666666666666, 0,0,0,0,0,0,0,0,0}, /* ros_C */
        {1,0,1,1,0,0,0,0}, /* ros_NewF */
        {2.0,0,1.0,1.0,0,0}, /* ros_M */
        {0,0,0,1.0,0,0}, /* ros_E */
        {0,0,1.0,1.0,0,0}, /* ros_Alpha */
        {0.5,1.5,0,0,0,0},  /* ros_Gamma */
        3.0, /* ros_ELO */
        4
    }, /* Rodas3 */

    { 
        {
            0.1544000000000000E+01,  0.9466785280815826E+00, 0.2557011698983284E+00, 0.3314825187068521E+01,
            0.2896124015972201E+01,  0.9986419139977817E+00, 0.1221224509226641E+01, 0.6019134481288629E+01,
            0.1253708332932087E+02, -0.6878860361058950E+00, 0.1221224509226641E+01, 0.6019134481288629E+01,
            0.1253708332932087E+02, -0.6878860361058950E+00, 1.0E+00},  /* ros_A */ 

        {
            -0.5668800000000000E+01, -0.2430093356833875E+01, -0.2063599157091915E+00, -0.1073529058151375E+00,  
            -0.9594562251023355E+01, -0.2047028614809616E+02,  0.7496443313967647E+01, -0.1024680431464352E+02,  
            -0.3399990352819905E+02,  0.1170890893206160E+02,  0.8083246795921522E+01, -0.7981132988064893E+01,  
            -0.3152159432874371E+02,  0.1631930543123136E+02, -0.6058818238834054E+01}, /* ros_C */
        {1,1,1,1,1,1,0,0}, /* ros_NewF */
        {0.1221224509226641E+01,0.6019134481288629E+01,0.1253708332932087E+02,- 0.6878860361058950E+00,1,1}, /* ros_M */
        {0,0,0,0,0,1.0}, /* ros_E */
        {0.000,  0.386,  0.210,  0.630,  1.000, 1.000}, /* ros_Alpha */
        {0.2500000000000000E+00,  -0.1043000000000000E+00,  0.1035000000000000E+00,  0.3620000000000023E-01, 0, 0},  /* ros_Gamma */
        4.0, /* ros_ELO */
        6
    } /* Rodas4 */



};



//__device__ double rconst_local[MAX_VL_GLO*NREACT];

/* Initialize rconst local  */
//__device__ double * rconst_local;


__device__ double k_3rd(double temp, double cair, double k0_300K, double n, double kinf_300K, double m, double fc)
    /*
 *    
 * temp        temperature [K]
 * cair        air concentration [molecules/cm3]
 * k0_300K     low pressure limit at 300 K
 * n           exponent for low pressure limit
 * kinf_300K   high pressure limit at 300 K
 * m           exponent for high pressure limit
 * fc          broadening factor (usually fc=0.6)
 * 
 */
{

    double zt_help, k0_T, kinf_T, k_ratio, k_3rd_r;

    zt_help = 300.0/temp;
    k0_T    = k0_300K   *pow(zt_help,n) *cair;
    kinf_T  = kinf_300K *pow(zt_help,m);
    k_ratio = k0_T/kinf_T;
    k_3rd_r   = k0_T/(1.0+ k_ratio)*pow(fc,1.0/(1.0+ pow(log10(k_ratio),2)));
    return k_3rd_r;
}

__device__ double k_3rd_iupac(double temp, double cair, double k0_300K, double n, double kinf_300K, double m, double fc)
/*
 *    
 * temp        temperature [K]
 * cair        air concentration [molecules/cm3]
 * k0_300K     low pressure limit at 300 K
 * n           exponent for low pressure limit
 * kinf_300K   high pressure limit at 300 K
 * m           exponent for high pressure limit
 * fc          broadening factor (e.g. 0.45 or 0.6...)
 * nu          N
 * 
 */
{   
 
    double zt_help, k0_T, kinf_T, k_ratio, nu, k_3rd_iupac_r;
    zt_help = 300.0/temp;
    k0_T    = k0_300K   *pow(zt_help,n) *cair;
    kinf_T  = kinf_300K *pow(zt_help,m);
    k_ratio = k0_T/kinf_T;
    nu      = 0.75- 1.27*log10(fc);
    k_3rd_iupac_r = k0_T/(1.0+ k_ratio)*pow(fc,1.0/(1.0+ pow(log10(k_ratio)/nu,2)));
    return k_3rd_iupac_r;
}




double * temp_gpu;
double * press_gpu;
double * cair_gpu;


=#=#=#=#=#=#=#=#=#=#=update_rconst=#=#=#=#=#=#=#=#=#=#=


__global__ 
void Rosenbrock(double * __restrict__ conc, const double Tstart, const double Tend, double * __restrict__ rstatus, int * __restrict__ istatus,
                // values calculated from icntrl and rcntrl at host
                const int autonomous, const int vectorTol, const int UplimTol, const int method, const int Max_no_steps,
                double * __restrict__ d_jac0, double * __restrict__ d_Ghimj, double * __restrict__ d_varNew, double * __restrict__ d_K, double * __restrict__ d_varErr,double * __restrict__ d_dFdT ,double * __restrict__ d_Fcn0, double * __restrict__ d_var, double * __restrict__ d_fix, double * __restrict__ d_rconst,
                const double Hmin, const double Hmax, const double Hstart, const double FacMin, const double FacMax, const double FacRej, const double FacSafe, const double roundoff,
                // cuda global mem buffers              
                const double * __restrict__ absTol, const double * __restrict__ relTol,
                // for update_rconst
    	        const double * __restrict__ khet_st, const double * __restrict__ khet_tr,
		const double * __restrict__ jx,
                // global input
                const double * __restrict__ temp_gpu,
                const double * __restrict__ press_gpu,
                const double * __restrict__ cair_gpu,
                // extra
                const int VL_GLO,
								// Offset for stream computation
                const int offset
)
{
    int index = blockIdx.x*blockDim.x+threadIdx.x+offset;

    /* 
     *  In theory someone can aggregate accesses together,
     *  however due to algorithm, threads access 
     *  different parts of memory, making it harder to
     *  optimize accesses. 
     *
     */

    if (index < VL_GLO)
    {

    double *Ghimj  = &d_Ghimj[index*LU_NONZERO];    
    double *K      = &d_K[index*NVAR*6];
    double *varNew = &d_varNew[index*NVAR];
    double *Fcn0   = &d_Fcn0[index*NVAR];
    double *dFdT   = &d_dFdT[index*NVAR];
    double *jac0   = &d_jac0[index*LU_NONZERO];
    double *varErr = &d_varErr[index*NVAR];
    double *var    = &d_var[index*NSPEC];
    double *fix    = &d_fix[index*NFIX];
    double *rconst = &d_rconst[index*NREACT];



        int Nfun,Njac,Nstp,Nacc,Nrej,Ndec,Nsol,Nsng;
        double Texit, Hexit;

        Nfun = 0;
        Njac = 0;
        Nstp = 0;
        Nacc = 0;
        Nrej = 0;
        Ndec = 0;
        Nsol = 0;
        Nsng = 0;

        /* FIXME: add check for method */
        const double *ros_A     = &ros[method-1].ros_A[0]; 
        const double *ros_C     = &ros[method-1].ros_C[0];
        const double *ros_M     = &ros[method-1].ros_M[0]; 
        const double *ros_E     = &ros[method-1].ros_E[0];
        const double *ros_Alpha = &ros[method-1].ros_Alpha[0]; 
        const double *ros_Gamma = &ros[method-1].ros_Gamma[0]; 
        const int    *ros_NewF  = &ros[method-1].ros_NewF[0];
        const int     ros_S     =  ros[method-1].ros_S; 
        const double  ros_ELO   =  ros[method-1].ros_ELO; 





        /* Copy data from global memory to temporary array */
        /*
         * Optimization note: if we ever have enough constant
         * memory, we could use it for storing the data.
         * In current architectures if we use constant memory
         * only a few threads will be able to run on the fly.
         *
         */
        for (int i=0; i<NSPEC; i++)
            var(index,i) = conc(index,i);

        for (int i=0; i<NFIX; i++)
            fix(index,i) = conc(index,NVAR+i);


        update_rconst(var, khet_st, khet_tr, jx, rconst, temp_gpu, press_gpu, cair_gpu, VL_GLO, offset); 

        ros_Integrator(var, fix, Tstart, Tend, Texit,
                //  Rosenbrock method coefficients
                ros_S, ros_M, ros_E, ros_A, ros_C, 
                ros_Alpha, ros_Gamma, ros_ELO, ros_NewF, 
                //  Integration parameters
                autonomous, vectorTol, Max_no_steps, 
                roundoff, Hmin, Hmax, Hstart, Hexit, 
                FacMin, FacMax, FacRej, FacSafe,
                //  Status parameters
                Nfun, Njac, Nstp, Nacc, Nrej, Ndec, Nsol, Nsng,
                //  cuda global mem buffers              
                rconst, absTol, relTol, varNew, Fcn0,  
                K, dFdT, jac0, Ghimj,  varErr, 
                // For update rconst
                khet_st, khet_tr, jx,
                VL_GLO
                );

        for (int i=0; i<NVAR; i++)
            conc(index,i) = var(index,i); 


        /* Statistics */
        istatus(index,ifun) = Nfun;
        istatus(index,ijac) = Njac;
        istatus(index,istp) = Nstp;
        istatus(index,iacc) = Nacc;
        istatus(index,irej) = Nrej;
        istatus(index,idec) = Ndec;
        istatus(index,isol) = Nsol;
        istatus(index,isng) = Nsng;
        // Last T and H
        rstatus(index,itexit) = Texit;
        rstatus(index,ihexit) = Hexit; 
    }
}


=#=#=#=#=#=#=#=#=#=#=special_ros=#=#=#=#=#=#=#=#=#=#=


                                                        // no int8 in CUDA :(
__global__ void reduce_istatus_1(int *istatus, int4 *tmp_out_1, int4 *tmp_out_2, int VL_GLO, int *xNacc, int *xNrej)
{
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int idx_1 = threadIdx.x;
    int global_size = blockDim.x*gridDim.x;
    
    int foo;
    //no int8 in CUDA :(
    int4 accumulator_1 = make_int4(0,0,0,0);
    int4 accumulator_2 = make_int4(0,0,0,0);
    while (index < VL_GLO)
    {
        accumulator_1.x += istatus(index,0);
        accumulator_1.y += istatus(index,1);
        accumulator_1.z += istatus(index,2);
        //some dirty work on the side...
        foo = istatus(index,3);
        xNacc[index] = foo;
        accumulator_1.w += foo;
        foo = istatus(index,4);
        xNrej[index] = foo;
        accumulator_2.x += foo;
        accumulator_2.y += istatus(index,5);
        accumulator_2.z += istatus(index,6);
        accumulator_2.w += istatus(index,7);
        index += global_size;
    }
    //no int8 in CUDA :(
    __shared__ int4 buffer_1[REDUCTION_SIZE_1];
    __shared__ int4 buffer_2[REDUCTION_SIZE_1];
    
    buffer_1[idx_1] = accumulator_1;
    buffer_2[idx_1] = accumulator_2;
    __syncthreads();
    
    int idx_2, active_threads = blockDim.x;
    int4 tmp_1, tmp_2;
    while (active_threads != 1)
    {
        active_threads /= 2;
        if (idx_1 < active_threads)
        {
            idx_2 = idx_1+active_threads;
            
            tmp_1 = buffer_1[idx_1];
            tmp_2 = buffer_1[idx_2];
            
            tmp_1.x += tmp_2.x;
            tmp_1.y += tmp_2.y;
            tmp_1.z += tmp_2.z;
            tmp_1.w += tmp_2.w;
            
            buffer_1[idx_1] = tmp_1;
            
            
            tmp_1 = buffer_2[idx_1];
            tmp_2 = buffer_2[idx_2];
            
            tmp_1.x += tmp_2.x;
            tmp_1.y += tmp_2.y;
            tmp_1.z += tmp_2.z;
            tmp_1.w += tmp_2.w;
            
            buffer_2[idx_1] = tmp_1;
            
        }
        __syncthreads();
    }
    if (idx_1 == 0)
    {
        tmp_out_1[blockIdx.x] = buffer_1[0];
        tmp_out_2[blockIdx.x] = buffer_2[0];
    }
}            

__global__ void reduce_istatus_2(int4 *tmp_out_1, int4 *tmp_out_2, int *out)
{
    int idx_1 = threadIdx.x;
    //no int8 in CUDA :(
    __shared__ int4 buffer_1[REDUCTION_SIZE_2];
    __shared__ int4 buffer_2[REDUCTION_SIZE_2];
    
    buffer_1[idx_1] = tmp_out_1[idx_1];
    buffer_2[idx_1] = tmp_out_2[idx_1]; 
    __syncthreads();
    
    int idx_2, active_threads = blockDim.x;
    int4 tmp_1, tmp_2;
    while (active_threads != 1)
    {
        active_threads /= 2;
        if (idx_1 < active_threads)
        {
            idx_2 = idx_1+active_threads;
            
            tmp_1 = buffer_1[idx_1];
            tmp_2 = buffer_1[idx_2];
            
            tmp_1.x += tmp_2.x;
            tmp_1.y += tmp_2.y;
            tmp_1.z += tmp_2.z;
            tmp_1.w += tmp_2.w;
            
            buffer_1[idx_1] = tmp_1;
            
            
            tmp_1 = buffer_2[idx_1];
            tmp_2 = buffer_2[idx_2];
            
            tmp_1.x += tmp_2.x;
            tmp_1.y += tmp_2.y;
            tmp_1.z += tmp_2.z;
            tmp_1.w += tmp_2.w;
            
            buffer_2[idx_1] = tmp_1;
            
        }
        __syncthreads();
    }
    if (idx_1 == 0)
    {
        tmp_1 = buffer_1[0];
        tmp_2 = buffer_2[0];
        out[0] = tmp_1.x;
        out[1] = tmp_1.y;
        out[2] = tmp_1.z;
        out[3] = tmp_1.w;
        out[4] = tmp_2.x;
        out[5] = tmp_2.y;
        out[6] = tmp_2.z;
        out[7] = tmp_2.w;        
    }
}            

/* Assuming different processes */
enum { TRUE=1, FALSE=0 } ;
double *d_conc, *d_khet_st, *d_khet_tr, *d_jx, *d_jac0, *d_Ghimj, *d_varNew, *d_K, *d_varErr, *d_dFdT, *d_Fcn0, *d_var, *d_fix, *d_rconst;
double *h_conc, *h_temp, *h_press, *h_cair, *h_khet_st, *h_khet_tr, *h_jx, *h_absTol, *h_relTol;
int initialized = FALSE;

/* Device pointers pointing to GPU */
double *d_rstatus, *d_absTol, *d_relTol;
int *d_istatus, *d_istatus_rd, *d_xNacc, *d_xNrej;
int4 *d_tmp_out_1, *d_tmp_out_2;

/* number of streams, blocks per kernel call, and threads per block */
int nStreams;
int streamSize;
int nBlocks;
hipStream_t *stream;

/* Allocate arrays on device for Rosenbrock */
__host__ void init_first_time(int pe, int VL_GLO, int size_khet_st, int size_khet_tr, int size_jx ){

    /* Select the proper GPU CARD */
    int deviceCount, device;
    gpuErrchk( hipGetDeviceCount(&deviceCount) );
    device = pe % deviceCount;
    gpuErrchk( hipSetDevice(device) );

    printf("PE[%d]: selected %d of total %d\n",pe,device,deviceCount);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    gpuErrchk( hipMalloc ((void **) &d_conc   , sizeof(double)*VL_GLO*(NSPEC))        );
    gpuErrchk( hipMalloc ((void **) &d_khet_st, sizeof(double)*VL_GLO*size_khet_st) );
    gpuErrchk( hipMalloc ((void **) &d_khet_tr, sizeof(double)*VL_GLO*size_khet_tr) );
    gpuErrchk( hipMalloc ((void **) &d_jx     , sizeof(double)*VL_GLO*size_jx)      );

    gpuErrchk( hipMalloc ((void **) &d_rstatus    , sizeof(double)*VL_GLO*2)          );
    gpuErrchk( hipMalloc ((void **) &d_istatus    , sizeof(int)*VL_GLO*8)             );
    gpuErrchk( hipMalloc ((void **) &d_absTol     , sizeof(double)*NVAR)              );
    gpuErrchk( hipMalloc ((void **) &d_relTol     , sizeof(double)*NVAR)              );

    /* Allocate input arrays */
    gpuErrchk( hipMalloc ((void **) &temp_gpu     , sizeof(double)*VL_GLO)              );
    gpuErrchk( hipMalloc ((void **) &press_gpu     , sizeof(double)*VL_GLO)              );
    gpuErrchk( hipMalloc ((void **) &cair_gpu     , sizeof(double)*VL_GLO)              );

    /* Allocate arrays on device for reducing metrics */
    gpuErrchk( hipMalloc ((void **) &d_istatus_rd  , sizeof(int)*8));
    gpuErrchk( hipMalloc ((void **) &d_tmp_out_1   , sizeof(int4)*64));
    gpuErrchk( hipMalloc ((void **) &d_tmp_out_2   , sizeof(int4)*64));
    gpuErrchk( hipMalloc ((void **) &d_xNacc   , sizeof(int)*VL_GLO));
    gpuErrchk( hipMalloc ((void **) &d_xNrej   , sizeof(int)*VL_GLO));

    /* Allocate arrays for solvers on device global memory to reduce the stack usage */
    gpuErrchk( hipMalloc ((void **) &d_jac0, sizeof(double)*VL_GLO*LU_NONZERO)   );
    gpuErrchk( hipMalloc ((void **) &d_Ghimj, sizeof(double)*VL_GLO*LU_NONZERO)   );
    gpuErrchk( hipMalloc ((void **) &d_varNew, sizeof(double)*VL_GLO*NVAR)       );
    gpuErrchk( hipMalloc ((void **) &d_Fcn0, sizeof(double)*VL_GLO*NVAR)       );
    gpuErrchk( hipMalloc ((void **) &d_dFdT, sizeof(double)*VL_GLO*NVAR)       );

    gpuErrchk( hipMalloc ((void **) &d_K, sizeof(double)*VL_GLO*NVAR*6)       );  // TODO: Change size according to solver steps
    gpuErrchk( hipMalloc ((void **) &d_varErr, sizeof(double)*VL_GLO*NVAR)       );
    gpuErrchk( hipMalloc ((void **) &d_var, sizeof(double)*VL_GLO*NSPEC)       );
    gpuErrchk( hipMalloc ((void **) &d_fix, sizeof(double)*VL_GLO*NFIX)       );
    gpuErrchk( hipMalloc ((void **) &d_rconst, sizeof(double)*VL_GLO*NREACT)       );

    /* Allocate Staging area #TODO if the arrays would already be in pinnend memory
     * that would be much better */
    int grid_size = (VL_GLO + BLOCKSIZE - 1)/BLOCKSIZE;  
    hipHostMalloc((void**) &h_conc    , sizeof(double)*VL_GLO*(NSPEC));
    hipHostMalloc((void**) &h_temp    , sizeof(double)*VL_GLO);
    hipHostMalloc((void**) &h_press   , sizeof(double)*VL_GLO);
    hipHostMalloc((void**) &h_cair    , sizeof(double)*VL_GLO);
    hipHostMalloc((void**) &h_khet_st , sizeof(double)*VL_GLO*size_khet_st);
    hipHostMalloc((void**) &h_khet_tr , sizeof(double)*VL_GLO*size_khet_tr);
    hipHostMalloc((void**) &h_jx      , sizeof(double)*VL_GLO*size_jx);
    hipHostMalloc((void**) &h_absTol  , sizeof(double)*NVAR);
    hipHostMalloc((void**) &h_relTol  , sizeof(double)*NVAR);

    nStreams = grid_size;
    nBlocks = (grid_size-1)/nStreams+1;
    streamSize = nBlocks*BLOCKSIZE;
    //recalulate nstreams
    nStreams = (grid_size-1)/nBlocks+1;
    stream = (hipStream_t*)malloc(sizeof(hipStream_t)*nStreams); 
    for (int i = 0; i < nStreams; ++i) {
      gpuErrchk( hipStreamCreate(&stream[i]) );
    }

#ifdef DEBUG 
    printf("CUDA Setup:\n");
    printf("  grid_size: %d\n",grid_size);
    printf("  block_size: %d\n",BLOCKSIZE);
    printf("  nStreams: %d\n",nStreams);
    printf("  nBlocks_per_stream: %d\n",nBlocks);
    printf("  streamSize: %d\n",streamSize);
    printf("  VL_GLO: %d\n",VL_GLO);
#endif

    initialized = TRUE;
}

/*
 * TODO: We should call it in some point..
 */
extern "C" void finalize_cuda(){
    /* Free memory on the device */
    gpuErrchk( hipFree(d_conc        ) );
    gpuErrchk( hipFree(d_khet_st     ) );
    gpuErrchk( hipFree(d_khet_tr     ) );
    gpuErrchk( hipFree(d_jx          ) );
    gpuErrchk( hipFree(d_rstatus     ) );
    gpuErrchk( hipFree(d_istatus     ) );
    gpuErrchk( hipFree(d_absTol      ) );
    gpuErrchk( hipFree(d_relTol      ) );
    gpuErrchk( hipFree(d_istatus_rd  ) );
    gpuErrchk( hipFree(d_tmp_out_1   ) );
    gpuErrchk( hipFree(d_tmp_out_2   ) );
    gpuErrchk( hipFree(d_xNacc       ) );
    gpuErrchk( hipFree(d_xNrej       ) );
    gpuErrchk( hipFree(temp_gpu      ) );
    gpuErrchk( hipFree(press_gpu     ) );
    gpuErrchk( hipFree(cair_gpu      ) );

    gpuErrchk( hipFree(d_jac0        ) );
    gpuErrchk( hipFree(d_Ghimj       ) );
    gpuErrchk( hipFree(d_varNew      ) );
    gpuErrchk( hipFree(d_Fcn0        ) );
    gpuErrchk( hipFree(d_dFdT        ) );
    gpuErrchk( hipFree(d_K           ) );
    gpuErrchk( hipFree(d_varErr      ) );
    gpuErrchk( hipFree(d_var         ) );
    gpuErrchk( hipFree(d_fix         ) );
    gpuErrchk( hipFree(d_rconst      ) );

    for (int i = 0; i < nStreams; ++i) {
      gpuErrchk( hipStreamDestroy(stream[i]) );
    }
    free(stream);
}



extern "C" void kpp_integrate_cuda_( int *pe_p, int *sizes, double *time_step_len_p, double *conc, double *temp, double *press, double *cair, 
                                    double *khet_st, double *khet_tr, double *jx, double *absTol, double *relTol, int *ierr, int *istatus, 
                                    int *xNacc, int *xNrej, double *rndoff, int *icntrl=NULL, double *rcntrl=NULL
				    ) 
/*  // TODO
 *  Parameters:
 *          pe_p: scalar int - processor element
 *        VL_GLO: scalar int - size of the system
 *         NSPEC: scalar int - number of species
 *        NREACT: scalar int - number of reactions
 *          NVAR: scalar int - 
 *
 *  Input data:
 *          conc: 2D array of doubles - size: vl_glo x number of species
 *          temp: 1D array of doubles - size: vl_glo
 *         press: 1D array of doubles - size: vl_glo
 *          cair: 1D array of doubles - size: vl_glo
 *       khet_st: 2D array of doubles - size: vl_glo x number of species
 *       khet_tr: 2D array of doubles - size: vl_glo x number of species 
 *            jx: 2D array of doubles - size: vl_glo x number of species
 *        absTol: 1D array of doubles - size: number of species
 *        relTol: 1D array of doubles - size: number of species
 *     Control:
 *        icntrl: 1D array of ints   - size: 4
 *         sizes: 1D array of ints   - size: 4
 *        rcntrl: 1D array of doubles - size: 7
 * 
 * 
 */
{

    const double DELTAMIN = 1.0E-5;

    
    int VL_GLO       = sizes[0];
    int size_khet_st = sizes[1];
    int size_khet_tr = sizes[2];
    int size_jx      = sizes[3];
    double roundoff  = *rndoff; 
    
    double Tstart,Tend;
    Tstart = ZERO;
    Tend   =  *time_step_len_p;
    int pe = *pe_p;
    
    // variables from rcntrl and icntrl
    int autonomous, vectorTol, UplimTol, method, Max_no_steps;
    double Hmin, Hmax, Hstart, FacMin, FacMax, FacRej, FacSafe;
    
    //int rcntrl_bool = 0, icntrl_bool=0;
    if (rcntrl == NULL)
    {
        rcntrl = new double[7];
        for (int i=0; i < 7; i++)
            rcntrl[i] = 0.0;
    }
    if (icntrl == NULL)
    {
        icntrl = new int[4];
        for (int i=0; i < 4; i++)
            icntrl[i] = 0;
    }

    /* Allocate arrays on device for update_rconst kernel*/        
    if (initialized == FALSE)   init_first_time(pe, VL_GLO, size_khet_st, size_khet_tr, size_jx);

    /* Compute execution configuration for update_rconst */
    int block_size, grid_size;
    
    block_size = BLOCKSIZE;
    grid_size = (VL_GLO + block_size - 1)/block_size;  
    dim3 dimBlock(block_size);

//  *------------------------------------------------------*
//  |    Default values vs input settings (icntrl, rcntrl) |
//  *------------------------------------------------------*
    int ierr_tmp=0;
    {
    //  autonomous or time dependent ODE. Default is time dependent.
        autonomous = !(icntrl[0] == 0);

    //  For Scalar tolerances (icntrl[1].NE.0)  the code uses absTol(0) and relTol(0)
    //  For Vector tolerances (icntrl[1] == 0) the code uses absTol(0:NVAR) and relTol(0:NVAR)
        if (icntrl[1] == 0)
        {
            vectorTol = 1; //bool
            UplimTol  = NVAR;
        }
        else
        {
            vectorTol = 0;
            UplimTol  = 1;
        }

    //  The particular Rosenbrock method chosen
        if (icntrl[2] == 0) 
        {
            method = 4;
        }
        else if ((icntrl[2] >= 1) && (icntrl[2] <= 5))
        {
            method = icntrl[2];
        }
        else
        {
            printf("User-selected Rosenbrock method: icntrl[2]=%d\n",method);
            ierr_tmp = -2;
        }
    //  The maximum number of steps admitted
        if (icntrl[3] == 0)
        {
            Max_no_steps = 100000;
        }
        else if (icntrl[3] > 0) 
        {
            Max_no_steps=icntrl[3];
        }
        else
        {
            printf("User-selected max no. of steps: icntrl[3]=%d\n",icntrl[3]);
            ierr_tmp = -1;
        }
    //  Unit roundoff (1+ roundoff>1)
        roundoff = machine_eps_flt(); 

    //  Lower bound on the step size: (positive value)
        if (rcntrl[0] == ZERO)
        {
            Hmin = ZERO;
        }
        else if (rcntrl[0] > ZERO) 
        {
            Hmin = rcntrl[0];
        }
        else
        {
            printf("User-selected Hmin: rcntrl[0]=%f\n",rcntrl[0]);
            ierr_tmp = -3;
        }
    //  Upper bound on the step size: (positive value)
        if (rcntrl[1] == ZERO) 
        {
            Hmax = fabs(Tend-Tstart);
        }
        else if (rcntrl[1] > ZERO) 
        {
            Hmax = fmin(fabs(rcntrl[1]),fabs(Tend-Tstart));
        }
        else
        {
            printf("User-selected Hmax: rcntrl[1]=%f\n",rcntrl[1]);
            ierr_tmp = -3;
        }
    //  Starting step size: (positive value)
        if (rcntrl[2] == ZERO) 
        {
            Hstart = fmax(Hmin,DELTAMIN);
        }
        else if (rcntrl[2] > ZERO) 
        {
            Hstart = fmin(fabs(rcntrl[2]),fabs(Tend-Tstart));
        }
        else
        {
            printf("User-selected Hstart: rcntrl[2]=%f\n",rcntrl[2]);
            ierr_tmp = -3;
        }
    //  Step size can be changed s.t.  FacMin < Hnew/Hexit < FacMax
        if (rcntrl[3] == ZERO)
        {
            FacMin = 0.2;
        }
        else if (rcntrl[3] > ZERO) 
        {
            FacMin = rcntrl[3];
        }
        else
        {
            printf("User-selected FacMin: rcntrl[3]=%f\n",rcntrl[3]);
            ierr_tmp = -4;
        }
        if (rcntrl[4] == ZERO) 
        {
            FacMax = 6.0;
        }
        else if (rcntrl[4] > ZERO) 
        {
            FacMax = rcntrl[4];
        }
        else
        {
            printf("User-selected FacMax: rcntrl[4]=%f\n",rcntrl[4]);
            ierr_tmp = -4; 
        }
    //  FacRej: Factor to decrease step after 2 succesive rejections
        if (rcntrl[5] == ZERO) 
        {
            FacRej = 0.1;
        }
        else if (rcntrl[5] > ZERO) 
        {
            FacRej = rcntrl[5];
        }
        else
        {
            printf("User-selected FacRej: rcntrl[5]=%f\n",rcntrl[5]);
            ierr_tmp = -4;
        }
    //  FacSafe: Safety Factor in the computation of new step size
        if (rcntrl[6] == ZERO) 
        {
            FacSafe = 0.9;
        }
        else if (rcntrl[6] > ZERO)
        {
            FacSafe = rcntrl[6];
        }
        else
        {
            printf("User-selected FacSafe: rcntrl[6]=%f\n",rcntrl[6]);
            ierr_tmp = -4;
        }
    //  Check if tolerances are reasonable
        for (int i=0; i < UplimTol; i++)
        {
            if ((absTol[i] <= ZERO) || (relTol[i] <= 10.0*roundoff) || (relTol[i] >= 1.0))
            {
                printf("CCC absTol(%d) = %f \n",i,absTol[i]);
                printf("CCC relTol(%d) = %f \n",i,relTol[i]);
                ierr_tmp = -5;
            }
        }
    }


    memcpy(h_absTol, absTol  , sizeof(double)*NVAR);
    memcpy(h_relTol, relTol  , sizeof(double)*NVAR);
    /* Copy arrays from host memory to device memory for Rosenbrock */    
    gpuErrchk( hipMemcpy(d_absTol, h_absTol, sizeof(double)*NVAR, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_relTol, h_relTol, sizeof(double)*NVAR, hipMemcpyHostToDevice) );

    /* Copy to staging area */
    memcpy(h_conc, conc, sizeof(double)*VL_GLO*NSPEC);
    memcpy(h_temp, temp, sizeof(double)*VL_GLO);
    memcpy(h_press, press, sizeof(double)*VL_GLO);
    memcpy(h_cair, cair, sizeof(double)*VL_GLO);
    memcpy(h_khet_st, khet_st, sizeof(double)*VL_GLO*size_khet_st);
    memcpy(h_khet_tr, khet_tr, sizeof(double)*VL_GLO*size_khet_tr);
    memcpy(h_jx, jx, sizeof(double)*VL_GLO*size_jx);

    dim3 dimGrid(nBlocks);
    for (int i = 0; i < nStreams; ++i) {
      int offset = i * streamSize;
      // Dont read beyond what we have
      int size = min(streamSize,VL_GLO-offset);

      /* Copy data to staging area and subequently to device
			 #TODO: allocate memory on host, so that this is
       * not necessary TODO: put in concurrent loop */
      gpuErrchk( hipMemcpyAsync(&d_conc[offset*NSPEC], &h_conc[offset*NSPEC], sizeof(double)*size*NSPEC, hipMemcpyHostToDevice, stream[i]) );

      gpuErrchk( hipMemcpyAsync(&temp_gpu[offset]  , &h_temp[offset]    , sizeof(double)*size  , hipMemcpyHostToDevice,stream[i]));
      gpuErrchk( hipMemcpyAsync(&press_gpu[offset] , &h_press[offset]   , sizeof(double)*size  , hipMemcpyHostToDevice,stream[i]));
      gpuErrchk( hipMemcpyAsync(&cair_gpu[offset]  , &h_cair[offset]    , sizeof(double)*size , hipMemcpyHostToDevice,stream[i] ));

      gpuErrchk( hipMemcpyAsync(&d_khet_st[offset*size_khet_st] , &h_khet_st[offset*size_khet_st] , sizeof(double)*size*size_khet_st , hipMemcpyHostToDevice, stream[i] ));
      gpuErrchk( hipMemcpyAsync(&d_khet_tr[offset*size_khet_tr] , &h_khet_tr[offset*size_khet_tr] , sizeof(double)*size*size_khet_tr , hipMemcpyHostToDevice, stream[i] ));
      gpuErrchk( hipMemcpyAsync(&d_jx[offset*size_jx]      , &h_jx[offset*size_jx]      , sizeof(double)*size*size_jx  , hipMemcpyHostToDevice, stream[i] ));

    /* Execute the kernel */
    //update_rconst<<<dimGrid,dimBlock>>>(d_conc, d_khet_st, d_khet_tr, d_jx, VL_GLO); 
}
hipDeviceSynchronize();
    for (int i = 0; i < nStreams; ++i) {
      int offset = i * streamSize;
 
    =#=#=#=#=#=#=#=#=#=#=call_kernel=#=#=#=#=#=#=#=#=#=#=

		}
hipDeviceSynchronize();

    reduce_istatus_1<<<REDUCTION_SIZE_2,REDUCTION_SIZE_1>>>(d_istatus, d_tmp_out_1, d_tmp_out_2, VL_GLO, d_xNacc, d_xNrej);


    GPU_DEBUG();

    reduce_istatus_2<<<1,REDUCTION_SIZE_2>>>(d_tmp_out_1, d_tmp_out_2, d_istatus_rd);

    GPU_DEBUG();
    
    /* Copy the result back */
    gpuErrchk( hipMemcpy( conc      , d_conc       , sizeof(double)*VL_GLO*NVAR, hipMemcpyDeviceToHost) );  
    gpuErrchk( hipMemcpy( xNacc      , d_xNacc      , sizeof(int)*VL_GLO         , hipMemcpyDeviceToHost) );  
    gpuErrchk( hipMemcpy( xNrej      , d_xNrej      , sizeof(int)*VL_GLO         , hipMemcpyDeviceToHost) ); 

    
    return;

}





